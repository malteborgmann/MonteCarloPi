#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// CUDA-Kernel zur Initialisierung von cuRAND
__global__ void init_curand(hiprandState *state, unsigned long seed, long long num_points) {
    long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_points) {
        // Initialisiere den cuRAND Zustand mit dem Seed, der Thread-ID und der Position
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}

// CUDA-Kernel zur Generierung und Überprüfung von Punkten mit cuRAND
__global__ void monte_carlo_pi_kernel(long long num_points, hiprandState *state, unsigned long long *count) {
    extern __shared__ unsigned long long shared_count[];
    unsigned int tid = threadIdx.x;
    shared_count[tid] = 0;
    __syncthreads();

    long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long stride = blockDim.x * gridDim.x;

    for (long long i = idx; i < num_points; i += stride) {
        // Generiere zufällige x und y zwischen -1 und 1
        double x = 2.0 * hiprand_uniform_double(&state[i]) - 1.0;
        double y = 2.0 * hiprand_uniform_double(&state[i]) - 1.0;

        // Überprüfe, ob der Punkt innerhalb des Kreises liegt
        if (x * x + y * y <= 1.0) {
            shared_count[tid]++;
        }
    }

    __syncthreads();

    // Reduziere die lokalen Zähler innerhalb des Blocks
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_count[tid] += shared_count[tid + s];
        }
        __syncthreads();
    }

    // Atomare Addition nur einmal pro Block
    if (tid == 0) {
        atomicAdd(count, shared_count[0]);
    }
}

int main(int argc, char *argv[]) {
    // Überprüfen, ob die Anzahl der Punkte als Argument übergeben wurde
    if (argc != 2) {
        printf("Usage: %s <number_of_points>\n", argv[0]);
        return 1;
    }

    // Anzahl der Punkte aus den Kommandozeilenargumenten lesen
    long long num_points = atoll(argv[1]);
    if (num_points <= 0) {
        printf("Bitte geben Sie eine positive Anzahl von Punkten ein.\n");
        return 1;
    }

    // Host-Zähler initialisieren
    unsigned long long h_count = 0;

    // Device-Zähler initialisieren
    unsigned long long *d_count;
    hipError_t err = hipMalloc((void **)&d_count, sizeof(unsigned long long));
    if (err != hipSuccess) {
        printf("CUDA Error (hipMalloc): %s\n", hipGetErrorString(err));
        return 1;
    }
    err = hipMemcpy(d_count, &h_count, sizeof(unsigned long long), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA Error (hipMemcpy H2D): %s\n", hipGetErrorString(err));
        hipFree(d_count);
        return 1;
    }

    // cuRAND-Zustände initialisieren
    hiprandState *d_state;
    err = hipMalloc((void **)&d_state, num_points * sizeof(hiprandState));
    if (err != hipSuccess) {
        printf("CUDA Error (hipMalloc d_state): %s\n", hipGetErrorString(err));
        hipFree(d_count);
        return 1;
    }

    // Anzahl der Threads und Blocks festlegen
    int threads_per_block = 256;
    int blocks = (num_points + threads_per_block - 1) / threads_per_block;
    // Begrenze die Anzahl der Blocks, um die maximale Anzahl zu vermeiden
    blocks = min(blocks, 1024);

    // Seed für cuRAND
    unsigned long seed = (unsigned long)time(NULL);

    // Startzeit für die Messung der Laufzeit mit CUDA-Ereignissen
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Initialisiere cuRAND-Zustände
    init_curand<<<blocks, threads_per_block>>>(d_state, seed, num_points);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error (init_curand): %s\n", hipGetErrorString(err));
        hipFree(d_count);
        hipFree(d_state);
        return 1;
    }
    hipDeviceSynchronize();

    // Starte den Monte-Carlo-Kernel mit cuRAND
    size_t shared_mem_size = threads_per_block * sizeof(unsigned long long);
    monte_carlo_pi_kernel<<<blocks, threads_per_block, shared_mem_size>>>(num_points, d_state, d_count);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error (monte_carlo_pi_kernel): %s\n", hipGetErrorString(err));
        hipFree(d_count);
        hipFree(d_state);
        return 1;
    }
    hipDeviceSynchronize();

    // Endzeit nach der Berechnung
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_spent_ms;
    hipEventElapsedTime(&time_spent_ms, start, stop);

    // Ergebnis vom Device zum Host kopieren
    err = hipMemcpy(&h_count, d_count, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("CUDA Error (hipMemcpy D2H): %s\n", hipGetErrorString(err));
        hipFree(d_count);
        hipFree(d_state);
        return 1;
    }

    // Schätzung von π berechnen
    double pi_estimate = 4.0 * ((double)h_count / (double)num_points);

    // Ergebnisse ausgeben
    printf("Anzahl der Punkte: %lld\n", num_points);
    printf("Punkte innerhalb des Kreises: %llu\n", h_count);
    printf("Geschätzter Wert von π: %.10f\n", pi_estimate);
    printf("Abweichung von tatsächlichem π: %.10f\n", fabs(M_PI - pi_estimate));
    printf("Laufzeit: %.5f Sekunden\n", time_spent_ms / 1000.0f); // CUDA gibt Zeit in Millisekunden

    // Speicher freigeben
    hipFree(d_count);
    hipFree(d_state);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
