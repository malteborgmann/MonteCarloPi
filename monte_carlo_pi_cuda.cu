#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256
#define GRID_SIZE  256
// Punkte pro Thread - kann an gewünschte Genauigkeit angepasst werden
#define POINTS_PER_THREAD 10000

__global__ void setup_kernel(hiprandState *state, unsigned long long seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void pi_kernel(hiprandState *state, unsigned long long *countInside, long long pointsPerThread) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = state[id];

    unsigned long long inside = 0ULL;
    for (long long i = 0; i < pointsPerThread; i++) {
        float x = hiprand_uniform(&localState) * 2.0f - 1.0f;
        float y = hiprand_uniform(&localState) * 2.0f - 1.0f;
        float dist = x*x + y*y;
        if (dist <= 1.0f) {
            inside++;
        }
    }

    state[id] = localState;
    atomicAdd(countInside, inside);
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <number_of_points>\n", argv[0]);
        return 1;
    }

    long long num_points = atoll(argv[1]);
    if (num_points <= 0) {
        printf("Number of Points.\n");
        return 1;
    }

    // Anzahl Threads gesamt
    int totalThreads = BLOCK_SIZE * GRID_SIZE;
    // Berechne wie viele Punkte jeder Thread erzeugen soll
    // Falls num_points nicht durch totalThreads teilbar ist, wird abgerundet.
    long long pointsPerThread = num_points / totalThreads;
    if (pointsPerThread <= 0) {
        // Falls zu wenige Punkte für die gewählte Thread-Konfiguration:
        // Einfach mindestens 1 Punkt pro Thread testen.
        pointsPerThread = 1;
    }

    // Timer für Laufzeitmessung
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hiprandState *d_state;
    hipMalloc(&d_state, totalThreads * sizeof(hiprandState));

    unsigned long long *d_countInside;
    hipMalloc(&d_countInside, sizeof(unsigned long long));
    hipMemset(d_countInside, 0, sizeof(unsigned long long));

    // Startzeit messen
    hipEventRecord(start, 0);

    // RNG initialisieren
    setup_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_state, 1234ULL);
    hipDeviceSynchronize();

    // Kernel zum Zählen der Punkte innerhalb des Kreises
    pi_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_state, d_countInside, pointsPerThread);
    hipDeviceSynchronize();

    // Ende der Berechnung
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Zeit auslesen
    float time_spent = 0.0f;
    hipEventElapsedTime(&time_spent, start, stop);

    unsigned long long h_countInside = 0ULL;
    hipMemcpy(&h_countInside, d_countInside, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    double pi_estimate = 4.0 * (double)h_countInside / (double)(pointsPerThread * totalThreads);

    printf("Anzahl der Punkte: %lld\n", (long long)(pointsPerThread * totalThreads));
    printf("Punkte innerhalb des Kreises: %llu\n", h_countInside);
    printf("Geschätzter Wert von π: %.10f\n", pi_estimate);
    printf("Abweichung von tatsächlichem π: %.10f\n", fabs(M_PI - pi_estimate));
    // CUDA Event-Zeit ist in Millisekunden, Umrechnung in Sekunden
    printf("Laufzeit: %.5f Sekunden\n", time_spent / 1000.0f);

    hipFree(d_state);
    hipFree(d_countInside);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
